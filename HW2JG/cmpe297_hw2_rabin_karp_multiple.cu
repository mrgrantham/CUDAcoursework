#include "hip/hip_runtime.h"
// CMPE297-6 HW2
// CUDA version Rabin-Karp

#include<stdio.h>
#include<iostream>

#define NUM_PATTERN 4
#define PATTERN_MAX_LENGTH 15
/*ADD CODE HERE: Implement the parallel version of the sequential Rabin-Karp*/
__global__ void 
findIfExistsCu(char* input, int input_length, char* pattern, int *pattern_length, int *patHash, char* result, unsigned long long* runtime)
{
    //printf("Starting kernal thread %d block %d with pattern start %d\n",threadIdx.x,blockIdx.x,pattern_length[threadIdx.x]);
    
    unsigned long long start_time = clock64();
    int loc_in_input = threadIdx.x;
    int pattern_num = blockIdx.x; 
    int input_hash,i;

    //printf("input section: ");
    for(input_hash=0,i=loc_in_input;i<pattern_length[pattern_num]+loc_in_input;i++) {
        //printf("%c",input[i]);
        input_hash=((input_hash << 8) + input[i]) % 997;
    }
    //printf("\n");
    int j;
    bool match = true;
    __syncthreads();
    if (input_hash == patHash[pattern_num]) {
        int pattern_start_index = 0;
        for(int pat=0;pat < blockIdx.x;pat++) {
            pattern_start_index += pattern_length[pat];
        } 
        for(j=loc_in_input;j<loc_in_input+pattern_length[pattern_num];j++) {
           //printf("Block %d Thread %d [%d] pattern char %c input char %c\n",blockIdx.x,threadIdx.x,j,pattern[j-loc_in_input],input[j]);
           match &= (pattern[pattern_start_index+(j-loc_in_input)] == input[j]);
        }
        result[pattern_num] |= match?1:0;
    }

    if(result[pattern_num]){
        //printf("***pattern found in kernel at [%d]\n",pattern_num);
    }else {
       //////////////// printf("nothing here\n");
    }
	unsigned long long stop_time = clock64();
    runtime[(input_length*pattern_num)+loc_in_input] = (unsigned long long)(stop_time-start_time);
}

int main()
{
	// host variables
	char input[] = "Searching for multiple patterns in the input sequence."; 	/*Sample Input*/
	const char *pattern[NUM_PATTERN] = {"multiple","s i","ddd","seq"}; 		/*Sample Pattern*/
    char *patternflat;
    int patHash[NUM_PATTERN]; 			/*hash for the pattern*/
	char* result; 				/*Result array*/
	int pattern_length[NUM_PATTERN];		/*Pattern Length*/
	int input_length = strlen(input); 		/*Input Length*/
	// device variables
	char* d_input;
	char* d_pattern;
	char* d_result;
    int * d_pattern_length;
    int * d_patHash;
    
    int patternflat_length=0;
	/*Calculate the hash of the pattern*/
    for(int pl = 0; pl < NUM_PATTERN;pl++) {
        pattern_length[pl]=strlen(pattern[pl]);
        patternflat_length += pattern_length[pl];
    }
    patternflat = (char*)malloc(patternflat_length * sizeof(char));
    int flatindex = 0;
    for(int p = 0;p < NUM_PATTERN;p++) {
        //printf("strlen for \"%s\" is %d\n",pattern[p],pattern_length[p]);	
        memcpy(patternflat+flatindex,pattern[p],pattern_length[p]);
        flatindex += pattern_length[p];
        for (int i = 0; i < pattern_length[p]; i++)
        {
            patHash[p] = (patHash[p] * 256 + pattern[p][i]) % 997;
        }
   
    }
    //printf("\ndone calculating hash\n");
	// measure the execution time by using clock() api in the kernel as we did in Lab3
	int runtime_size = input_length*NUM_PATTERN*sizeof(unsigned long long);

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
	
    unsigned long long* d_runtime;
	result = (char *) malloc((NUM_PATTERN)*sizeof(char));
    memset(result,0,NUM_PATTERN*sizeof(char));
	unsigned long long* runtime = (unsigned long long *) malloc(runtime_size);
	memset(runtime,0,runtime_size);
    err = hipMalloc((void **)&d_runtime,runtime_size);
    
    if (err != hipSuccess) {
        fprintf(stderr,"Failed to allocated d_runtime (error code %s)!\n",hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
	/*ADD CODE HERE: Allocate memory on the GPU and copy or set the appropriate values from the HOST*/
    
	
    err = hipMalloc((void **)&d_input,input_length);
    if (err != hipSuccess) {
        fprintf(stderr,"Failed to allocated input (error code %s)!\n",hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **)&d_pattern,flatindex*sizeof(char));
    if (err != hipSuccess) {
        fprintf(stderr,"Failed to allocated input (error code %s)!\n",hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **)&d_pattern_length,NUM_PATTERN*sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr,"Failed to allocated pattern_length[] (error code %s)!\n",hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    err = hipMalloc((void **)&d_patHash,NUM_PATTERN*sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr,"Failed to allocated patHash[] (error code %s)!\n",hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **)&d_result,(NUM_PATTERN)*sizeof(char));
    if (err != hipSuccess) {
        fprintf(stderr,"Failer to allocate result (error code %s)!\n",hipGetErrorString(err));
    }
    err = hipMemcpy(d_result,result,NUM_PATTERN*sizeof(char),hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr,"Failed to copy result (error code %s)!\n",hipGetErrorString(err));
    }
    
    err = hipMemcpy(d_pattern,patternflat,flatindex*sizeof(char),hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr,"Failed to copy pattern (error code %s)!\n",hipGetErrorString(err));
    }

    err = hipMemcpy(d_pattern_length,pattern_length,NUM_PATTERN*sizeof(int),hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr,"Failed to copy pattern_length[] (error code %s)!\n",hipGetErrorString(err));
    }
    
    err = hipMemcpy(d_patHash,patHash,NUM_PATTERN*sizeof(int),hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr,"Failed to copy payHash[] (error code %s)!\n",hipGetErrorString(err));
    }

    err = hipMemcpy(d_input,input,input_length,hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr,"Failed to copy input (error code %s)!\n",hipGetErrorString(err));
    }

	/*ADD CODE HERE: Launch the kernel and pass the arguments*/
    int blocks = NUM_PATTERN;
    int threads = input_length;
	findIfExistsCu<<<blocks,threads>>>(d_input,input_length,d_pattern,d_pattern_length,d_patHash,d_result,d_runtime);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr,"Failed to launch kernel (error code %s)!\n",hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(result,d_result,NUM_PATTERN*sizeof(char),hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr,"Failed to copy back result (error code %s)!\n",hipGetErrorString(err)); 
        exit(EXIT_FAILURE);
    }	
    hipDeviceSynchronize();	
	/*ADD CODE HERE: Copy the execution times from the GPU memory to HOST Code*/		


    hipMemcpy(runtime, d_runtime, runtime_size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    unsigned long long elapsed_time = 0;
    for(int i = 0; i < input_length*NUM_PATTERN; i++)
        if(elapsed_time < runtime[i])
            elapsed_time = runtime[i];

	printf("\nTotal cycles: %d \n", (int)elapsed_time);

    printf("Searching for multiple patterns in the input sequence\n");
    printf("Input string = %s\n",input);
    for (int i = 0;i < NUM_PATTERN;i++) {
        printf("Pattern: \"%s\" %s\n",pattern[i],result[i]?"was found":"was not found");
    }	
	/*ADD CODE HERE: COPY the result and print the result as in the HW description*/
	
	return 0;
}

