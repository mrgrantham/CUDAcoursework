#include "hip/hip_runtime.h"
// CMPE297-6 HW2
// CUDA version Rabin-Karp

#include<stdio.h>
#include<iostream>


/*ADD CODE HERE: Implement the parallel version of the sequential Rabin-Karp*/
__global__ void 
findIfExistsCu(char* input, int input_length, char* pattern, int pattern_length, int patHash, int* result)
{ 
	
	
}

int main()
{
	// host variables
	char input[] = "HEABAL"; 	/*Sample Input*/
	char pattern[] = "AB"; 		/*Sample Pattern*/
	int patHash = 0; 			/*hash for the pattern*/
	int* result; 				/*Result array*/
	int* runtime; 				/*Exection cycles*/
	int pattern_length = 2;		/*Pattern Length*/
	int input_length = 6; 		/*Input Length*/

	// device variables
	char* d_input;
	char* d_pattern;
	int* d_result;
	int* d_runtime;

	// measure the execution time by using clock() api in the kernel as we did in Lab3
	int runtime_size = /*FILL CODE HERE*/;

	result = (int *) malloc((input_length-pattern_length)*sizeof(int));
	runtime = (int *) malloc(runtime_size);
	
	/*Calculate the hash of the pattern*/
	for (int i = 0; i < M; i++)
    {
        patHash = (patHash * 256 + pattern[i]) % 997;
    }

	/*ADD CODE HERE: Allocate memory on the GPU and copy or set the appropriate values from the HOST*/

	
	/*ADD CODE HERE: Launch the kernel and pass the arguments*/
		
		
	/*ADD CODE HERE: Copy the execution times from the GPU memory to HOST Code*/		
	
	
	/*RUN TIME calculation*/
    unsigned long long elapsed_time = 0;
    for(int i = 0; i < input_length-pattern_length; i++)
        if(elapsed_time < runtime[i])
            elapsed_time = runtime[i];

    printf("Kernel Execution Time: %llu cycles\n", elapsed_time);
	printf("Total cycles: %d \n", elapsed_time);
	printf("Kernel Execution Time: %d cycles\n", elapsed_time);

	
	/*ADD CODE HERE: COPY the result and print the result as in the HW description*/
	
	return 0;
}

