#include "hip/hip_runtime.h"
// CMPE297-6 HW2
// CUDA version Rabin-Karp

#include<stdio.h>
#include<iostream>


/*ADD CODE HERE: Implement the parallel version of the sequential Rabin-Karp*/
__global__ void 
findIfExistsCu(char* input, int input_length, char* pattern, int pattern_length, int patHash, int* result, unsigned long long* runtime)
{
    unsigned long long start_time = clock64();
    int loc_in_input = threadIdx.x; 
    int input_hash,i;
    for(input_hash=0,i=loc_in_input;i<pattern_length+loc_in_input;i++) {
        input_hash=((input_hash << 8) + input[i]) % 997;
    } 
    int j;
    bool match = true;
    if (input_hash == patHash) {
        for(j=loc_in_input;j<loc_in_input+pattern_length;j++) {
           //printf("Thread %d [%d] pattern char %c input char %c\n",threadIdx.x,j,pattern[j-loc_in_input],input[j]);
           match &= (pattern[j-loc_in_input] == input[j]);
        }
        result[loc_in_input]= match?1:0;
    }
	unsigned long long stop_time = clock64();
    runtime[loc_in_input] = (unsigned long long)(stop_time-start_time);
}

int main()
{
	// host variables
	char input[] = "HEABAL"; 	/*Sample Input*/
	char pattern[] = "AB"; 		/*Sample Pattern*/
	int patHash = 0; 			/*hash for the pattern*/
	int* result; 				/*Result array*/
	int* runtime; 				/*Exection cycles*/
	int pattern_length = 2;		/*Pattern Length*/
	int input_length = 6; 		/*Input Length*/

	// device variables
	char* d_input;
	char* d_pattern;
	int* d_result;
	unsigned long long* d_runtime;

	// measure the execution time by using clock() api in the kernel as we did in Lab3
	int runtime_size = input_length-pattern_length+1;

	result = (int *) malloc((input_length-pattern_length+1)*sizeof(int));
	runtime = (int *) malloc(runtime_size);
	memset(runtime,0,input_length-pattern_length+1);
    hipMalloc((void **)&d_runtime,input_length-pattern_length+1);
	/*Calculate the hash of the pattern*/
	for (int i = 0; i < pattern_length; i++)
    {
        patHash = (patHash * 256 + pattern[i]) % 997;
    }

	/*ADD CODE HERE: Allocate memory on the GPU and copy or set the appropriate values from the HOST*/
    
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
	
    err = hipMalloc((void **)&d_input,input_length);
    if (err != hipSuccess) {
        fprintf(stderr,"Failed to allocated input (error code %s)!\n",hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **)&d_pattern,pattern_length);
    if (err != hipSuccess) {
        fprintf(stderr,"Failed to allocated input (error code %s)!\n",hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **)&d_result,(input_length-pattern_length+1)*sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr,"Failer to allocate result (error code %s)!\n",hipGetErrorString(err));
    }

    err = hipMemcpy(d_pattern,pattern,pattern_length,hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr,"Failed to copy pattern (error code %s)!\n",hipGetErrorString(err));
    }

    err = hipMemcpy(d_input,input,input_length,hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr,"Failed to copy input (error code %s)!\n",hipGetErrorString(err));
    }

	/*ADD CODE HERE: Launch the kernel and pass the arguments*/
    int blocks = 1;
    int threads = input_length-pattern_length+1;
	findIfExistsCu<<<blocks,threads>>>(d_input,input_length,d_pattern,pattern_length,patHash,d_result,d_runtime);

    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr,"Failed to launch kernel (error code %s)!\n",hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(result,d_result,(input_length-pattern_length+1)*sizeof(int),hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr,"Failed to copy result (error code %s)!\n",hipGetErrorString(err)); 
        exit(EXIT_FAILURE);
    }	
    hipDeviceSynchronize();	
	/*ADD CODE HERE: Copy the execution times from the GPU memory to HOST Code*/		


    hipMemcpy(runtime, d_runtime, runtime_size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    unsigned long long elapsed_time = 0;
    for(int i = 0; i < input_length-pattern_length+1; i++)
        if(elapsed_time < runtime[i])
            elapsed_time = runtime[i];

	printf("Total cycles: %d \n", (int)elapsed_time);

    printf("Searching for a single pattern in a single string\n");
    printf("Print at which position the pattern was found\n");
    printf("Input string = %s\n",input);
    printf("pattern=%s\n",pattern);
    for (int i = 0;i < input_length-pattern_length+1;i++) {
        printf("Pos:%d Result: %d\n",i,result[i]);
    }	
	/*ADD CODE HERE: COPY the result and print the result as in the HW description*/
	
	return 0;
}

